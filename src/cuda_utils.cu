#include <iostream>

#include <hip/hip_runtime.h>


void checkCudaErrors(hipError_t result) {
    if (result != hipSuccess) {
        std::cerr << "CUDA Runtime Error: " << hipGetErrorString(result) << std::endl;
        throw std::runtime_error("CUDA Runtime Error");
    }
}